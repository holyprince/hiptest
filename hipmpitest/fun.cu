#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>

#include <mpi.h>

#include <hip/hip_runtime.h>

#define VECTOR_SIZE 0x10000
#define BLOCK_SIZE 512

__global__ void vec_add(double *a, double *b, double *c, int n)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if (id < n)
		c[id] = a[id] + b[id];
}

void do_compute(void)
{
	double *A_h, *B_h, *C_h;
	double *A_d, *B_d, *C_d;
	int size = VECTOR_SIZE;
	int nb = size * sizeof(double);

	A_h = (double *)malloc(nb);
	B_h = (double *)malloc(nb);
	C_h = (double *)malloc(nb);
	assert(A_h && B_h && C_h);
  
	for (int i = 0; i < size; i++) {
		A_h[i] = B_h[i] = 1;
		C_h[i] = 0;
	}

	assert(hipMalloc(&A_d, nb) == hipSuccess);
	assert(hipMalloc(&B_d, nb) == hipSuccess);
	assert(hipMalloc(&C_d, nb) == hipSuccess);

	assert(hipMemcpy(A_d, A_h, nb, hipMemcpyHostToDevice) == hipSuccess);
	assert(hipMemcpy(B_d, B_h, nb, hipMemcpyHostToDevice) == hipSuccess);

	hipLaunchKernelGGL(vec_add, dim3(VECTOR_SIZE/BLOCK_SIZE), dim3(BLOCK_SIZE),
			   0, 0, A_d, B_d, C_d, size);

	assert(hipMemcpy(C_h, C_d, nb, hipMemcpyDeviceToHost) == hipSuccess);
	for (int i = 0; i < size; i++)
		assert(C_h[i] == 2);

	free(A_h);
	free(B_h);
	free(C_h);
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
}

